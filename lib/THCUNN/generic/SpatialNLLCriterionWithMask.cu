
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/SpatialNLLCriterionWithMask.cu"
#else

void THNN_(SpatialNLLCriterionWithMask_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCIndexTensor *target,
           THCTensor *valid_mask,
           THCTensor *output,
           bool sizeAverage,
           THCTensor *weights,
           THCTensor *total_weight)
{
  THArgCheck(THCIndexTensor_(nDimension)(state, target) == 3, 1,
             "only batches of spatial targets supported (3D tensors)" \
             " but got targets of dimension: %d",
             THCIndexTensor_(nDimension)(state, target));
  THArgCheck(THCTensor_(nDimension)(state, input) == 4, 2,
             "only batches of spatial inputs supported (4D tensors), "      \
             "but got input of dimension: %d", THCTensor_(nDimension)(state, input));

  if (weights && THCTensor_(nElement)(state, weights) != THCTensor_(size)(state, input, 1)) {
    THError("weight tensor should be defined either for all or no classes");
  }

  if (weights)
    THCUNN_assertSameGPU(state, 5, input, target, weights, output, total_weight);
  else
    THCUNN_assertSameGPU(state, 4, input, target, output, total_weight);

  input = THCTensor_(newContiguous)(state, input);
  weights = weights ? THCTensor_(newContiguous)(state, weights) : NULL;
  target = THCIndexTensor_(newContiguous)(state, target);
  valid_mask = THCTensor_(newContiguous)(state, valid_mask);

  real *input_data = THCTensor_(data)(state, input);
  real *weights_data = weights ? THCTensor_(data)(state, weights) : NULL;
  THCIndex_t  *target_data = THCIndexTensor_(data)(state, target);
  real *valid_mask_data = THCTensor_(data)(state, valid_mask);
  real *output_data = THCTensor_(data)(state, output);
  real *total_weight_data = THCTensor_(data)(state, total_weight);

  THCIndex_t batch_size = THCIndexTensor_(size)(state, target, 0);
  THCIndex_t map_nelem = THCIndexTensor_(nElement)(state, target) / batch_size;
  int blocks_per_sample = GET_BLOCKS(map_nelem) / 128;
  blocks_per_sample = (blocks_per_sample == 0) ? 1 : blocks_per_sample;
  int total_blocks = blocks_per_sample * batch_size;

  THCTensor_(fill)(state, output, ScalarConvert<int, real>::to(0));
  THCTensor_(fill)(state, total_weight, ScalarConvert<int, real>::to(0));

  cunn_SpatialNLLCriterionWithMask_updateOutput_kernel<real, accreal>
    <<<total_blocks, CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state)>>>(
      output_data,
      total_weight_data,
      input_data,
      target_data,
      valid_mask_data,
      weights_data,
      sizeAverage,
      THCTensor_(size)(state, input, 0),
      THCTensor_(size)(state, input, 1),
      THCTensor_(size)(state, input, 2) * THCTensor_(size)(state, input, 3),
      blocks_per_sample
  );
  THCudaCheck(cudaGetLastError());

  if (weights)
    THCTensor_(free)(state, weights);
  THCIndexTensor_(free)(state, target);
  THCTensor_(free)(state, input);
  THCTensor_(free)(state, valid_mask);
}

void THNN_(SpatialNLLCriterionWithMask_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCIndexTensor *target,
           THCTensor *valid_mask,
           THCTensor *gradInput,
           bool sizeAverage,
           THCTensor *weights,
           THCTensor *total_weight)
{
  THArgCheck(THCIndexTensor_(nDimension)(state, target) == 3, 1,
             "only batches of spatial targets supported (3D tensors)");
  THArgCheck(THCTensor_(nDimension)(state, input) == 4, 2,
             "only batches of spatial inputs supported (4D tensors)");
  THArgCheck(THCTensor_(isContiguous)(state, gradInput), 4,
             "gradInput must be contiguous");
  if (weights && THCTensor_(nElement)(state, weights) != THCTensor_(size)(state, input, 1)) {
    THError("weight tensor should be defined either for all or no classes");
  }

  if (weights)
    THCUNN_assertSameGPU(state, 5, weights, input, target, gradInput, total_weight);
  else
    THCUNN_assertSameGPU(state, 4, input, target, gradInput, total_weight);

  input = THCTensor_(newContiguous)(state, input);
  weights = weights ? THCTensor_(newContiguous)(state, weights) : NULL;
  target = THCIndexTensor_(newContiguous)(state, target);
  valid_mask = THCTensor_(newContiguous)(state, valid_mask);

  real *weights_data = weights ? THCTensor_(data)(state, weights) : NULL;
  real *gradInput_data = THCTensor_(data)(state, gradInput);
  THCIndex_t *target_data = THCIndexTensor_(data)(state, target);
  real *valid_mask_data = THCTensor_(data)(state, valid_mask);
  real *total_weight_data = THCTensor_(data)(state, total_weight);

  THCIndex_t batch_size = THCIndexTensor_(size)(state, target, 0);
  THCIndex_t map_nelem = THCIndexTensor_(nElement)(state, target) / batch_size;
  int blocks_per_sample = GET_BLOCKS(map_nelem) / 128;
  blocks_per_sample = (blocks_per_sample == 0) ? 1 : blocks_per_sample;
  int total_blocks = blocks_per_sample * batch_size;

  cunn_SpatialNLLCriterionWithMask_updateGradInput_kernel
    <<<total_blocks, CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state)>>>(
      gradInput_data,
      target_data,
      valid_mask_data,
      weights_data,
      total_weight_data,
      sizeAverage,
      THCTensor_(size)(state, input, 0),
      THCTensor_(size)(state, input, 1),
      THCTensor_(size)(state, input, 2) *THCTensor_(size)(state, input, 3),
      blocks_per_sample
  );
  THCudaCheck(cudaGetLastError());

  if (weights)
    THCTensor_(free)(state, weights);
  THCIndexTensor_(free)(state, target);
  THCTensor_(free)(state, input);
  THCTensor_(free)(state, valid_mask);
}

#endif
